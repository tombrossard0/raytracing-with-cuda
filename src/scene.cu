#include "hip/hip_runtime.h"
#include <iomanip> // for std::setw, std::setfill
#include <iostream>

#include "imgui.h"
#include "imgui_impl_opengl3.h"
#include "imgui_impl_sdl2.h"
#include "ppm.hpp"
#include "render.cuh"
#include "scene.hpp"

#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <sstream>

#include <cstdlib> // for rand()
#include <ctime>   // for seeding

#include <assimp/Importer.hpp>
#include <assimp/postprocess.h>
#include <assimp/scene.h>
#include <vector>

void scene1(Entity *entities, int &nEntities, Camera *cam) {
    nEntities = 6;
    entities[0] =
        Entity(EntityType::SPHERE, cam->center + Vec3(-17.218, -13.568, -3.990), 11.07, Vec3(1, 1, 1));
    entities[0].material.emissionColour = Vec3(1, 1, 1);
    entities[0].material.emissionStrength = 1.f;

    entities[1] = Entity(EntityType::SPHERE, cam->center + Vec3(0.92, -0.71, -3), .73f, Vec3(0, 1, 0));
    entities[2] = Entity(EntityType::SPHERE, cam->center + Vec3(2.23, -0.81, -6.13), .88f, Vec3(0, 0, 1));
    entities[3] = Entity(EntityType::SPHERE, cam->center + Vec3(1.59, 23.14, -3.850), 23.05, Vec3(1, 1, 1));
    entities[4] = Entity(EntityType::SPHERE, cam->center + Vec3(0.16, -1.52, -1.07), 1, Vec3(1, 0, 0));
    entities[5] = Entity(EntityType::SPHERE, cam->center + Vec3(-2.3, -0.8, -2.69), 1, Vec3(1, 1, 0.45));
}

inline float randf() {
    return rand() / (float)RAND_MAX;
}

void scene2(Entity *entities, int &nEntities, Camera *cam) {
    nEntities = 10;
    srand((unsigned)time(0));
    // 2.5
    int k = 0;
    for (float i = -3.5f; i <= 3.5f; i += 3.5f) {
        for (float j = -3.5f; j <= 3.5f; j += 3.5f) {
            // if (i == 0 && j == 0) {
            //     entities[k] = Entity(cam->center + Vec3(0, -15.f, 0), 9.3, 1);
            //     entities[k].material.emissionColour = 1;
            //     entities[k++].material.emissionStrength = 3.f;
            //     continue;
            // }

            Vec3 randomColor(randf(), randf(), randf());

            if (abs(i) == abs(j)) {
                entities[k++] =
                    Entity(EntityType::SPHERE, cam->center + Vec3(i / 1.4f, 0, j / 1.4f), 1.f, randomColor);
                continue;
            }
            entities[k++] = Entity(EntityType::SPHERE, cam->center + Vec3(i, 0, j), 1.f, randomColor);
        }
    }

    entities[9] = Entity(EntityType::SPHERE, cam->center + Vec3(0, 26.f, 0), 25.f, 1);
}

void scene3(Entity *entities, int &nEntities, Camera *cam) {
    nEntities = 1;

    Triangle *triangles;
    hipMallocManaged(&triangles, 1 * sizeof(Triangle));

    float s = 1.0f; // half-size of the cube
    Vec3 center = cam->center;

    // Front face (+Z)
    triangles[0] = Triangle{center + Vec3(-s, -s, +s), center + Vec3(+s, -s, +s), center + Vec3(+s, +s, +s)};
    triangles[1] = Triangle{center + Vec3(-s, -s, +s), center + Vec3(+s, +s, +s), center + Vec3(-s, +s, +s)};

    // Back face (-Z)
    triangles[2] = Triangle{center + Vec3(+s, -s, -s), center + Vec3(-s, -s, -s), center + Vec3(-s, +s, -s)};
    triangles[3] = Triangle{center + Vec3(+s, -s, -s), center + Vec3(-s, +s, -s), center + Vec3(+s, +s, -s)};

    // Left face (-X)
    triangles[4] = Triangle{center + Vec3(-s, -s, -s), center + Vec3(-s, -s, +s), center + Vec3(-s, +s, +s)};
    triangles[5] = Triangle{center + Vec3(-s, -s, -s), center + Vec3(-s, +s, +s), center + Vec3(-s, +s, -s)};

    // Right face (+X)
    triangles[6] = Triangle{center + Vec3(+s, -s, +s), center + Vec3(+s, -s, -s), center + Vec3(+s, +s, -s)};
    triangles[7] = Triangle{center + Vec3(+s, -s, +s), center + Vec3(+s, +s, -s), center + Vec3(+s, +s, +s)};

    // Top face (+Y)
    triangles[8] = Triangle{center + Vec3(-s, +s, +s), center + Vec3(+s, +s, +s), center + Vec3(+s, +s, -s)};
    triangles[9] = Triangle{center + Vec3(-s, +s, +s), center + Vec3(+s, +s, -s), center + Vec3(-s, +s, -s)};

    // Bottom face (-Y)
    triangles[10] = Triangle{center + Vec3(-s, -s, -s), center + Vec3(+s, -s, -s), center + Vec3(+s, -s, +s)};
    triangles[11] = Triangle{center + Vec3(-s, -s, -s), center + Vec3(+s, -s, +s), center + Vec3(-s, -s, +s)};

    entities[0] = Entity(EntityType::MESH, 12, triangles); // 1 is material
}

void loadFBX(const std::string &path, std::vector<Triangle> &outTris) {
    Assimp::Importer importer;
    const aiScene *scene = importer.ReadFile(path, aiProcess_Triangulate | aiProcess_JoinIdenticalVertices |
                                                       aiProcess_PreTransformVertices);

    if (!scene || !scene->HasMeshes()) { throw std::runtime_error("Failed to load FBX file: " + path); }

    auto fixCoord = [](const aiVector3D &v) { return Vec3(v.x, -v.y, v.z); };

    for (unsigned int m = 0; m < scene->mNumMeshes; m++) {
        aiMesh *mesh = scene->mMeshes[m];
        for (unsigned int f = 0; f < mesh->mNumFaces; f++) {
            aiFace &face = mesh->mFaces[f];
            if (face.mNumIndices != 3) continue;

            aiVector3D v0 = mesh->mVertices[face.mIndices[0]];
            aiVector3D v1 = mesh->mVertices[face.mIndices[1]];
            aiVector3D v2 = mesh->mVertices[face.mIndices[2]];

            outTris.push_back(Triangle{fixCoord(v0), fixCoord(v2), fixCoord(v1)});
        }
    }
}

void scene4(Entity *entities, int &nEntities) {
    nEntities = 1;

    std::vector<Triangle> hostTriangles;
    loadFBX("models/Knight.fbx", hostTriangles);

    Triangle *triangles;
    hipMallocManaged(&triangles, hostTriangles.size() * sizeof(Triangle));
    memcpy(triangles, hostTriangles.data(), hostTriangles.size() * sizeof(Triangle));

    entities[0] = Entity(EntityType::MESH, hostTriangles.size(), triangles);
    entities[0].size = 0.01;
}

Scene::Scene(int w, int h) : width(w), height(h), fb(nullptr), entities(nullptr), nEntities(0), texture(0) {
    makeCamera();

    size_t fb_size = width * height * sizeof(Vec3);
    hipMallocManaged(&fb, fb_size);

    hipMallocManaged(&entities, MAX_ENTITIES * sizeof(Entity));
    // scene3(entities, nEntities, cam);
    scene4(entities, nEntities);
}

Scene::~Scene() {
    if (texture) { glDeleteTextures(1, &texture); }

    hipDeviceSynchronize(); // ensure all kernels are finished
    if (fb) { hipFree(fb); };
    if (entities) {
        for (int i = 0; i < nEntities; i++) { hipFree(entities[i].triangles); }
        hipFree(entities);
    };
    if (cam) { hipFree(cam); }
}

void Scene::makeCamera() {
    hipMallocManaged(&cam, sizeof(Camera));

    cam->radius = 15.0f;
    cam->yawDeg = 64.0f;
    cam->pitchDeg = -16.0f;
    cam->minRadius = 1.0f;
    cam->maxRadius = 20.0;

    cam->maxBounces = 10;
    cam->numberOfRayPerPixel = 10;

    cam->center = Vec3(0, 0, 0);

    cam->updateCameraPosition();

    cam->up = Vec3(0, 1, 0);
    cam->fov = 90.0f;
    cam->aspect = float(width) / float(height);
}

void Scene::renderFrame(int i, int j) {
    cam->updateCameraPosition();
    render(i, j);
}

void Scene::renderGUI(GLuint &tex) {
    ImGui::Begin("Render Scene");
    ImGui::Image((void *)(intptr_t)tex, ImVec2(width, height));
    if (ImGui::IsItemClicked()) focus = true;
    bool hovered = ImGui::IsItemHovered();
    ImGui::End();

    if (!hovered && ImGui::IsMouseClicked(ImGuiMouseButton_Left)) focus = false;

    ImGui::Begin("Camera Controls");
    ImGui::SliderFloat("Radius", &cam->radius, cam->minRadius, cam->maxRadius);
    ImGui::SliderFloat("Yaw", &cam->yawDeg, -180.0f, 180.0f);
    ImGui::SliderFloat("Pitch", &cam->pitchDeg, -89.0f, 89.0f);
    ImGui::DragFloat3("Center", &cam->center.x, 0.01f);
    ImGui::DragInt("Max Bounces", &cam->maxBounces, 1, 0, 1000);
    ImGui::DragInt("Number of ray per pixel", &cam->numberOfRayPerPixel, 1, 0, 1000);
    ImGui::End();

    ImGui::Begin("Spheres");
    if (ImGui::Button("Add Entity")) {
        if (nEntities < MAX_ENTITIES) { entities[nEntities++] = Entity(EntityType::SPHERE, Vec3(), 1); }
    }

    for (int i = 0; i < nEntities; i++) {
        std::string nodeLabel = "Entity " + std::to_string(i);
        if (ImGui::CollapsingHeader(nodeLabel.c_str())) {
            ImGui::DragFloat3(("Position##" + std::to_string(i)).c_str(), &entities[i].center.x, 0.01f);
            ImGui::DragFloat(("Size##" + std::to_string(i)).c_str(), &entities[i].size, 0.01f, 0.1f, 50.0f);
            ImGui::ColorEdit3(("Color##" + std::to_string(i)).c_str(), &entities[i].material.colour.x);
            ImGui::ColorEdit3(("Emission color##" + std::to_string(i)).c_str(),
                              &entities[i].material.emissionColour.x);
            ImGui::DragFloat(("EMission strength##" + std::to_string(i)).c_str(),
                             &entities[i].material.emissionStrength, 0.0f, 0.1f, 100.0f);

            if (ImGui::Button(("Remove##" + std::to_string(i)).c_str())) {
                for (int j = i; j < nEntities - 1; j++) entities[j] = entities[j + 1];
                nEntities--;
            }
        }
    }
    ImGui::End();

    ImGui::Begin("Screenshots");
    if (ImGui::Button("save PPM")) { renderPPM(); }
    ImGui::End();
}

void Scene::renderPPMFrame(const std::string &filename) {
    cam->numberOfRayPerPixel = 1000;
    render(0, 0);
    savePPM(filename, fb, width, height);
}

void Scene::renderPPM(const std::string &filename) {
    renderPPMFrame(filename);
    std::cout << "Static render saved to " << filename << std::endl;
}

void Scene::renderGIF(int nFrames, float totalAngle) {
    cam->pitchDeg = -90;
    for (int i = 0; i < nFrames; i++) {
        cam->yawDeg = (totalAngle / nFrames) * i;
        cam->updateCameraPosition();
        std::ostringstream filename;
        filename << "build/frame_" << std::setw(3) << std::setfill('0') << i << ".ppm";
        renderPPMFrame(filename.str());
        std::cout << "Saved " << filename.str() << std::endl;
    }
    std::cout << "Video render complete!" << std::endl;
}

void Scene::render(int numRenderedFramesA, int numRenderedFramesB) {
    dim3 threads(16, 16);
    dim3 blocks((width + 15) / 16, (height + 15) / 16);

    // Note: the kernel runs on the GPU, which cannot directly access host
    // memory unless we use managed memory or hipMemcpy
    SceneProperties sceneProperties{
        fb, width, height, entities, nEntities, cam, numRenderedFramesA, numRenderedFramesB};

    render_scene<<<blocks, threads>>>(sceneProperties);

    hipDeviceSynchronize();
}

void Scene::processInputs(InputManager inputManager, MouseState mouse, float deltaTime) {
    cam->processInputs(inputManager, mouse, deltaTime);
}
