#include <iomanip> // for std::setw, std::setfill
#include <iostream>

#include "imgui.h"
#include "imgui_impl_opengl3.h"
#include "imgui_impl_sdl2.h"
#include "ppm.hpp"
#include "render.hpp"
#include "scene.hpp"

#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <sstream>

Scene::Scene(int w, int h)
    : width(w), height(h), fb(nullptr), spheres(nullptr), nSpheres(0), radius(5.0f), yawDeg(0.0f),
      pitchDeg(0.0f), minRadius(1.0f), maxRadius(20.0), texture(0) {
    makeCamera();

    size_t fb_size = width * height * sizeof(Vec3);
    hipMallocManaged(&fb, fb_size);

    hipMallocManaged(&spheres, MAX_SPHERES * sizeof(Sphere));
    nSpheres = 4;
    spheres[0] = Sphere(cam->center + Vec3(-4.418, -5.648, -3), 5, Vec3(1, 1, 1));
    spheres[0].material.emissionColour = Vec3(1, 1, 1);
    spheres[0].material.emissionStrength = 1;

    spheres[1] = Sphere(cam->center + Vec3(0.92, 0, -3), .3f, Vec3(0, 1, 0));
    spheres[2] = Sphere(cam->center + Vec3(2.23, 1.05, -6.13), .4f, Vec3(0, 0, 1));
    spheres[3] = Sphere(cam->center + Vec3(1.59, 5.28, -3.850), 5, Vec3(1, 0, 0));
}

Scene::~Scene() {
    if (texture) { glDeleteTextures(1, &texture); }

    hipDeviceSynchronize(); // ensure all kernels are finished
    if (fb) { hipFree(fb); };
    if (spheres) { hipFree(spheres); };
    if (cam) { hipFree(cam); }
}

void Scene::makeCamera() {
    hipMallocManaged(&cam, sizeof(Camera));

    cam->maxBounces = 10;
    cam->numberOfRayPerPixel = 100;

    cam->center = Vec3(0, 0, 0);

    cam->updateCameraPosition(yawDeg, pitchDeg, radius);

    cam->up = Vec3(0, 1, 0);
    cam->fov = 90.0f;
    cam->aspect = float(width) / float(height);
}

void Scene::renderFrame() {
    render(fb, width, height, spheres, nSpheres, cam);
}

void Scene::renderGUI(GLuint &tex) {
    ImGui::Begin("Render Scene");
    ImGui::Image((void *)(intptr_t)tex, ImVec2(width, height));
    if (ImGui::IsItemClicked()) focus = true;
    bool hovered = ImGui::IsItemHovered();
    ImGui::End();

    if (!hovered && ImGui::IsMouseClicked(ImGuiMouseButton_Left)) focus = false;

    ImGui::Begin("Camera Controls");
    ImGui::SliderFloat("Radius", &radius, minRadius, maxRadius);
    ImGui::SliderFloat("Yaw", &yawDeg, -180.0f, 180.0f);
    ImGui::SliderFloat("Pitch", &pitchDeg, -89.0f, 89.0f);
    ImGui::DragFloat3("Center", &cam->center.x, 0.01f);
    ImGui::DragInt("Max Bounces", &cam->maxBounces, 1, 0, 1000);
    ImGui::DragInt("Number of ray per pixel", &cam->numberOfRayPerPixel, 1, 0, 1000);
    ImGui::End();

    ImGui::Begin("Spheres");
    if (ImGui::Button("Add Sphere")) {
        if (nSpheres < MAX_SPHERES) { spheres[nSpheres++] = Sphere(Vec3(), 1); }
    }

    for (int i = 0; i < nSpheres; i++) {
        std::string nodeLabel = "Sphere " + std::to_string(i);
        if (ImGui::CollapsingHeader(nodeLabel.c_str())) {
            ImGui::DragFloat3(("Position##" + std::to_string(i)).c_str(), &spheres[i].center.x, 0.01f);
            ImGui::DragFloat(("Radius##" + std::to_string(i)).c_str(), &spheres[i].radius, 0.01f, 0.1f,
                             50.0f);
            ImGui::ColorEdit3(("Color##" + std::to_string(i)).c_str(), &spheres[i].material.colour.x);
            ImGui::ColorEdit3(("Emission color##" + std::to_string(i)).c_str(),
                              &spheres[i].material.emissionColour.x);
            ImGui::DragFloat(("EMission strength##" + std::to_string(i)).c_str(),
                             &spheres[i].material.emissionStrength, 0.0f, 0.1f, 1.0f);

            if (ImGui::Button(("Remove##" + std::to_string(i)).c_str())) {
                for (int j = i; j < nSpheres - 1; j++) spheres[j] = spheres[j + 1];
                nSpheres--;
            }
        }
    }
    ImGui::End();

    ImGui::Begin("Screenshots");
    if (ImGui::Button("save PPM")) { renderPPM(); }
    ImGui::End();
}

void Scene::renderPPMFrame(const std::string &filename) {
    render(fb, width, height, spheres, nSpheres, cam);
    savePPM(filename, fb, width, height);
}

void Scene::renderPPM(const std::string &filename) {
    renderPPMFrame(filename);
    std::cout << "Static render saved to " << filename << std::endl;
}

void Scene::renderGIF(int nFrames, float totalAngle) {
    for (int i = 0; i < nFrames; i++) {
        yawDeg = (totalAngle / nFrames) * i;
        cam->updateCameraPosition(yawDeg, pitchDeg, radius);
        std::ostringstream filename;
        filename << "build/frame_" << std::setw(3) << std::setfill('0') << i << ".ppm";
        renderPPMFrame(filename.str());
        std::cout << "Saved " << filename.str() << std::endl;
    }
    std::cout << "Video render complete!" << std::endl;
}
