#include "hip/hip_runtime.h"
#include <iomanip> // for std::setw, std::setfill
#include <iostream>

#include "imgui.h"
#include "imgui_impl_opengl3.h"
#include "imgui_impl_sdl2.h"
#include "ppm.hpp"
#include "render.cuh"
#include "scene.hpp"

#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <iostream>
#include <sstream>

#include <cstdlib> // for rand()
#include <ctime>   // for seeding

void scene1(Sphere *spheres, int &nSpheres, Camera *cam) {
    nSpheres = 6;
    spheres[0] = Sphere(cam->center + Vec3(-17.218, -13.568, -3.990), 11.07, Vec3(1, 1, 1));
    spheres[0].material.emissionColour = Vec3(1, 1, 1);
    spheres[0].material.emissionStrength = 1;

    spheres[1] = Sphere(cam->center + Vec3(0.92, -0.71, -3), .73f, Vec3(0, 1, 0));
    spheres[2] = Sphere(cam->center + Vec3(2.23, -0.81, -6.13), .88f, Vec3(0, 0, 1));
    spheres[3] = Sphere(cam->center + Vec3(1.59, 23.14, -3.850), 23.05, Vec3(1, 1, 1));
    spheres[4] = Sphere(cam->center + Vec3(0.16, -1.52, -1.07), 1, Vec3(1, 0, 0));
    spheres[5] = Sphere(cam->center + Vec3(-2.3, -0.8, -2.69), 1, Vec3(1, 1, 0.45));
}

inline float randf() {
    return rand() / (float)RAND_MAX;
}

void scene2(Sphere *spheres, int &nSpheres, Camera *cam) {
    nSpheres = 10;
    srand((unsigned)time(0));
    // 2.5
    int k = 0;
    for (float i = -3.5f; i <= 3.5f; i += 3.5f) {
        for (float j = -3.5f; j <= 3.5f; j += 3.5f) {
            if (i == 0 && j == 0) {
                spheres[k] = Sphere(cam->center + Vec3(0, -15.f, 0), 9.3, 1);
                spheres[k].material.emissionColour = 1;
                spheres[k++].material.emissionStrength = 1;
                continue;
            }

            Vec3 randomColor(randf(), randf(), randf());

            if (abs(i) == abs(j)) {
                spheres[k++] = Sphere(cam->center + Vec3(i / 1.4f, 0, j / 1.4f), 1.f, randomColor);
                continue;
            }
            spheres[k++] = Sphere(cam->center + Vec3(i, 0, j), 1.f, randomColor);
        }
    }

    spheres[9] = Sphere(cam->center + Vec3(0, 26.f, 0), 25.f, 1);
}

Scene::Scene(int w, int h) : width(w), height(h), fb(nullptr), spheres(nullptr), nSpheres(0), texture(0) {
    makeCamera();

    size_t fb_size = width * height * sizeof(Vec3);
    hipMallocManaged(&fb, fb_size);

    hipMallocManaged(&spheres, MAX_SPHERES * sizeof(Sphere));
    scene2(spheres, nSpheres, cam);
}

Scene::~Scene() {
    if (texture) { glDeleteTextures(1, &texture); }

    hipDeviceSynchronize(); // ensure all kernels are finished
    if (fb) { hipFree(fb); };
    if (spheres) { hipFree(spheres); };
    if (cam) { hipFree(cam); }
}

void Scene::makeCamera() {
    hipMallocManaged(&cam, sizeof(Camera));

    cam->radius = 15.0f;
    cam->yawDeg = 64.0f;
    cam->pitchDeg = -16.0f;
    cam->minRadius = 1.0f;
    cam->maxRadius = 20.0;

    cam->maxBounces = 10;
    cam->numberOfRayPerPixel = 10;

    cam->center = Vec3(0, 0, 0);

    cam->updateCameraPosition();

    cam->up = Vec3(0, 1, 0);
    cam->fov = 90.0f;
    cam->aspect = float(width) / float(height);
}

void Scene::renderFrame(int i, int j) {
    cam->updateCameraPosition();
    render(i, j);
}

void Scene::renderGUI(GLuint &tex) {
    ImGui::Begin("Render Scene");
    ImGui::Image((void *)(intptr_t)tex, ImVec2(width, height));
    if (ImGui::IsItemClicked()) focus = true;
    bool hovered = ImGui::IsItemHovered();
    ImGui::End();

    if (!hovered && ImGui::IsMouseClicked(ImGuiMouseButton_Left)) focus = false;

    ImGui::Begin("Camera Controls");
    ImGui::SliderFloat("Radius", &cam->radius, cam->minRadius, cam->maxRadius);
    ImGui::SliderFloat("Yaw", &cam->yawDeg, -180.0f, 180.0f);
    ImGui::SliderFloat("Pitch", &cam->pitchDeg, -89.0f, 89.0f);
    ImGui::DragFloat3("Center", &cam->center.x, 0.01f);
    ImGui::DragInt("Max Bounces", &cam->maxBounces, 1, 0, 1000);
    ImGui::DragInt("Number of ray per pixel", &cam->numberOfRayPerPixel, 1, 0, 1000);
    ImGui::End();

    ImGui::Begin("Spheres");
    if (ImGui::Button("Add Sphere")) {
        if (nSpheres < MAX_SPHERES) { spheres[nSpheres++] = Sphere(Vec3(), 1); }
    }

    for (int i = 0; i < nSpheres; i++) {
        std::string nodeLabel = "Sphere " + std::to_string(i);
        if (ImGui::CollapsingHeader(nodeLabel.c_str())) {
            ImGui::DragFloat3(("Position##" + std::to_string(i)).c_str(), &spheres[i].center.x, 0.01f);
            ImGui::DragFloat(("Radius##" + std::to_string(i)).c_str(), &spheres[i].radius, 0.01f, 0.1f,
                             50.0f);
            ImGui::ColorEdit3(("Color##" + std::to_string(i)).c_str(), &spheres[i].material.colour.x);
            ImGui::ColorEdit3(("Emission color##" + std::to_string(i)).c_str(),
                              &spheres[i].material.emissionColour.x);
            ImGui::DragFloat(("EMission strength##" + std::to_string(i)).c_str(),
                             &spheres[i].material.emissionStrength, 0.0f, 0.1f, 1.0f);

            if (ImGui::Button(("Remove##" + std::to_string(i)).c_str())) {
                for (int j = i; j < nSpheres - 1; j++) spheres[j] = spheres[j + 1];
                nSpheres--;
            }
        }
    }
    ImGui::End();

    ImGui::Begin("Screenshots");
    if (ImGui::Button("save PPM")) { renderPPM(); }
    ImGui::End();
}

void Scene::renderPPMFrame(const std::string &filename) {
    cam->numberOfRayPerPixel = 1000;
    render(0, 0);
    savePPM(filename, fb, width, height);
}

void Scene::renderPPM(const std::string &filename) {
    renderPPMFrame(filename);
    std::cout << "Static render saved to " << filename << std::endl;
}

void Scene::renderGIF(int nFrames, float totalAngle) {
    cam->pitchDeg = -90;
    for (int i = 0; i < nFrames; i++) {
        cam->yawDeg = (totalAngle / nFrames) * i;
        cam->updateCameraPosition();
        std::ostringstream filename;
        filename << "build/frame_" << std::setw(3) << std::setfill('0') << i << ".ppm";
        renderPPMFrame(filename.str());
        std::cout << "Saved " << filename.str() << std::endl;
    }
    std::cout << "Video render complete!" << std::endl;
}

void Scene::render(int numRenderedFramesA, int numRenderedFramesB) {
    dim3 threads(16, 16);
    dim3 blocks((width + 15) / 16, (height + 15) / 16);

    // Note: the kernel runs on the GPU, which cannot directly access host
    // memory unless we use managed memory or hipMemcpy
    SceneProperties sceneProperties{
        fb, width, height, spheres, nSpheres, cam, numRenderedFramesA, numRenderedFramesB};

    render_scene<<<blocks, threads>>>(sceneProperties);

    hipDeviceSynchronize();
}

void Scene::processInputs(InputManager inputManager, MouseState mouse, float deltaTime) {
    cam->processInputs(inputManager, mouse, deltaTime);
}
